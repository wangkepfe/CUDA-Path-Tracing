#include "hip/hip_runtime.h"
/******************************************
 * 
 *          CUDA GPU path tracing
 * 
 * 
 * 
 * 
 * ****************************************/


// cuda
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// render kernal header
#include "CudaRenderKernel.h"

// c
#include "stdio.h"

// utils
#include "cutil_math.h"  // required for float3
#include "mymath.h"

// material modeling
#include "reflection.cuh"

// ******************* macro define ********************

// constants
#ifndef M_PI
#define M_PI 3.1415926535897932384626422832795028841971f
#endif
#define F32_MIN          (1.175494351e-38f)
#define F32_MAX          (3.402823466e+38f)

// bvh stack
#define STACK_SIZE  64  // Size of the traversal stack in local memory.
#define EntrypointSentinel 0x76543210

// limits
#define RAY_MIN 1e-5f
#define RAY_MAX 1e20f
#define M_EPSILON 1e-5f
#define SCENE_MAX 1e5f

// sampling settings
#define NUM_SAMPLE 1
#define USE_RUSSIAN false
#define RUSSIAN_P 0.98
#define LIGHT_BOUNCE 80

// ******************* structures ********************

// enum
enum Refl_t { MAT_EMIT, MAT_DIFF, MAT_MIRROR, MAT_GLASS, MAT_NO };  // material types
enum Geo_t { GEO_TRIANGLE, GEO_SPHERE, GEO_GROUND };  // geo types
enum Medium_t {MEDIUM_NO = -1, MEDIUM_TEST = 0};

// geometry, material
struct Ray {
	Vec3f orig;	// ray origin
	Vec3f dir;		// ray direction	
	__device__ Ray(Vec3f o_, Vec3f d_) : orig(o_), dir(d_) {}
};

struct Sphere {
	float rad;				// radius 
	Vec3f pos, emi, col;	// position, emission, color 
	Refl_t refl;			// reflection type (DIFFuse, SPECular, REFRactive)
	int medium;

	__device__ float intersect(const Ray &r) const { // returns distance, 0 if nohit 
		// ray/sphere intersection
		Vec3f op = pos - r.orig;   
		float t;
		float b = dot(op, r.dir);
		float disc = b*b - dot(op, op) + rad*rad; // discriminant of quadratic formula
		if (disc<0) return 0; else disc = sqrtf(disc);
		return (t = b - disc) > M_EPSILON ? t : ((t = b + disc) > M_EPSILON ? t : 0.0f);
	}
};

struct GroundPlane {
	// normal (0, 1, 0)
	float y;
	__device__ float intersect(const Ray &r) const { // returns distance, 0 if nohit 
		return abs(r.dir.y) > M_EPSILON ? ((y - r.orig.y) / r.dir.y) : 0.0f;
	}
};

struct MediumSS {
	Vec3f sigmaS;
	Vec3f sigmaA;
	float g;
	__device__ Vec3f getSigmaT() { return sigmaA + sigmaS; }
};

// ******************* global variables ********************

// bvh
texture<float4, 1, hipReadModeElementType> bvhNodesTexture;
texture<float4, 1, hipReadModeElementType> triWoopTexture;
texture<float4, 1, hipReadModeElementType> triDebugTexture;
texture<int, 1, hipReadModeElementType> triIndicesTexture;
texture<float2, 1, hipReadModeElementType> triUvTexture;
texture<float4, 1, hipReadModeElementType> triNormalTexture;

// hdr
texture<float4, hipTextureType2D, hipReadModeElementType> HDRtexture;

// color texture
texture<float4, hipTextureType2D, hipReadModeElementType> colorTexture;

// ******************* math util func ********************

__device__ inline Vec3f absmax3f(const Vec3f& v1, const Vec3f& v2) { return Vec3f(v1.x*v1.x > v2.x*v2.x ? v1.x : v2.x, v1.y*v1.y > v2.y*v2.y ? v1.y : v2.y, v1.z*v1.z > v2.z*v2.z ? v1.z : v2.z); }
__device__ __inline__ int   min_min(int a, int b, int c) { int v; asm("vmin.s32.s32.s32.min %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   min_max(int a, int b, int c) { int v; asm("vmin.s32.s32.s32.max %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   max_min(int a, int b, int c) { int v; asm("vmax.s32.s32.s32.min %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ int   max_max(int a, int b, int c) { int v; asm("vmax.s32.s32.s32.max %0, %1, %2, %3;" : "=r"(v) : "r"(a), "r"(b), "r"(c)); return v; }
__device__ __inline__ float fmin_fmin(float a, float b, float c) { return __int_as_float(min_min(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmin_fmax(float a, float b, float c) { return __int_as_float(min_max(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmax_fmin(float a, float b, float c) { return __int_as_float(max_min(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }
__device__ __inline__ float fmax_fmax(float a, float b, float c) { return __int_as_float(max_max(__float_as_int(a), __float_as_int(b), __float_as_int(c))); }

__device__ __inline__ float spanBeginKepler(float a0, float a1, float b0, float b1, float c0, float c1, float d){ return fmax_fmax(fminf(a0, a1), fminf(b0, b1), fmin_fmax(c0, c1, d)); }
__device__ __inline__ float spanEndKepler(float a0, float a1, float b0, float b1, float c0, float c1, float d)	{ return fmin_fmin(fmaxf(a0, a1), fmaxf(b0, b1), fmax_fmin(c0, c1, d)); }
__device__ __inline__ void swap2(int& a, int& b){ int temp = a; a = b; b = temp;}

// ******************* functions ********************

// intersectBVHandTriangles
__device__ void intersectBVHandTriangles(
	const float4 rayorig, 
	const float4 raydir,
	int& hitTriIdx, 
	float& hitdistance, 
	Vec3f& trinormal,
	bool anyHit)
{
	// assign a CUDA thread to every pixel by using the threadIndex
	// global threadId, see richiesams blogspot
	//int thread_index = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	///////////////////////////////////////////
	//// KEPLER KERNEL
	///////////////////////////////////////////

	// BVH layout Compact2 for Kepler
	int traversalStack[STACK_SIZE];

	// Live state during traversal, stored in registers.

	//int		rayidx;		// not used, can be removed
	float   origx, origy, origz;    // Ray origin.
	float   dirx, diry, dirz;       // Ray direction.
	float   tmin;                   // t-value from which the ray starts. Usually 0.
	float   idirx, idiry, idirz;    // 1 / ray direction
	float   oodx, oody, oodz;       // ray origin / ray direction

	char*   stackPtr;               // Current position in traversal stack.
	int     leafAddr;               // If negative, then first postponed leaf, non-negative if no leaf (innernode).
	int     nodeAddr;
	int     hitIndex;               // Triangle index of the closest intersection, -1 if none.
	float   hitT;                   // t-value of the closest intersection.
	
	//int threadId1; // ipv rayidx

	// Initialize (stores local variables in registers)
	{
		// Pick ray index.

		//threadId1 = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (blockIdx.x + gridDim.x * blockIdx.y));
		

		// Fetch ray.
		origx = rayorig.x;
		origy = rayorig.y;
		origz = rayorig.z;
		dirx = raydir.x;
		diry = raydir.y;
		dirz = raydir.z;
		tmin = rayorig.w;

		// ooeps is very small number, used instead of raydir xyz component when that component is near zero
		float ooeps = exp2f(-80.0f); // Avoid div by zero, returns 1/2^80, an extremely small number
		idirx = 1.0f / (fabsf(raydir.x) > ooeps ? raydir.x : copysignf(ooeps, raydir.x)); // inverse ray direction
		idiry = 1.0f / (fabsf(raydir.y) > ooeps ? raydir.y : copysignf(ooeps, raydir.y)); // inverse ray direction
		idirz = 1.0f / (fabsf(raydir.z) > ooeps ? raydir.z : copysignf(ooeps, raydir.z)); // inverse ray direction
		oodx = origx * idirx;  // ray origin / ray direction
		oody = origy * idiry;  // ray origin / ray direction
		oodz = origz * idirz;  // ray origin / ray direction

		// Setup traversal + initialisation

		traversalStack[0] = EntrypointSentinel; // Bottom-most entry. 0x76543210 (1985229328 in decimal)
		stackPtr = (char*)&traversalStack[0]; // point stackPtr to bottom of traversal stack = EntryPointSentinel
		leafAddr = 0;   // No postponed leaf.
		nodeAddr = 0;   // Start from the root.
		hitIndex = -1;  // No triangle intersected so far.
		hitT = raydir.w; // tmax  
	}

	// Traversal loop.

	while (nodeAddr != EntrypointSentinel) 
	{
		// Traverse internal nodes until all SIMD lanes have found a leaf.

		//bool searchingLeaf = true; // required for warp efficiency
		while (nodeAddr >= 0 && nodeAddr != EntrypointSentinel)  
		{
			// Fetch AABBs of the two child nodes.

			// nodeAddr is an offset in number of bytes (char) in gpuNodes array
			
			float4 n0xy = tex1Dfetch(bvhNodesTexture, nodeAddr); // childnode 0, xy-bounds (c0.lo.x, c0.hi.x, c0.lo.y, c0.hi.y)		
			float4 n1xy = tex1Dfetch(bvhNodesTexture, nodeAddr + 1); // childnode 1, xy-bounds (c1.lo.x, c1.hi.x, c1.lo.y, c1.hi.y)		
			float4 nz = tex1Dfetch(bvhNodesTexture, nodeAddr + 2); // childnode 0 and 1, z-bounds (c0.lo.z, c0.hi.z, c1.lo.z, c1.hi.z)		
            float4 tmp = tex1Dfetch(bvhNodesTexture, nodeAddr + 3); // contains indices to 2 childnodes in case of innernode, see below
            int2 cnodes = *(int2*)&tmp; // cast first two floats to int
            // (childindex = size of array during building, see CudaBVH.cpp)

			// compute ray intersections with BVH node bounding box

			/// RAY BOX INTERSECTION
			// Intersect the ray against the child nodes.

			float c0lox = n0xy.x * idirx - oodx; // n0xy.x = c0.lo.x, child 0 minbound x
			float c0hix = n0xy.y * idirx - oodx; // n0xy.y = c0.hi.x, child 0 maxbound x
			float c0loy = n0xy.z * idiry - oody; // n0xy.z = c0.lo.y, child 0 minbound y
			float c0hiy = n0xy.w * idiry - oody; // n0xy.w = c0.hi.y, child 0 maxbound y
			float c0loz = nz.x   * idirz - oodz; // nz.x   = c0.lo.z, child 0 minbound z
			float c0hiz = nz.y   * idirz - oodz; // nz.y   = c0.hi.z, child 0 maxbound z
			float c1loz = nz.z   * idirz - oodz; // nz.z   = c1.lo.z, child 1 minbound z
			float c1hiz = nz.w   * idirz - oodz; // nz.w   = c1.hi.z, child 1 maxbound z
			float c0min = spanBeginKepler(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, tmin); // Tesla does max4(min, min, min, tmin)
			float c0max = spanEndKepler(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, hitT); // Tesla does min4(max, max, max, tmax)
			float c1lox = n1xy.x * idirx - oodx; // n1xy.x = c1.lo.x, child 1 minbound x
			float c1hix = n1xy.y * idirx - oodx; // n1xy.y = c1.hi.x, child 1 maxbound x
			float c1loy = n1xy.z * idiry - oody; // n1xy.z = c1.lo.y, child 1 minbound y
			float c1hiy = n1xy.w * idiry - oody; // n1xy.w = c1.hi.y, child 1 maxbound y
			float c1min = spanBeginKepler(c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, tmin);
			float c1max = spanEndKepler(c1lox, c1hix, c1loy, c1hiy, c1loz, c1hiz, hitT);

			// ray box intersection boundary tests:
			
			//float ray_tmax = 1e20;
			bool traverseChild0 = (c0min <= c0max); // && (c0min >= tmin) && (c0min <= ray_tmax);
			bool traverseChild1 = (c1min <= c1max); // && (c1min >= tmin) && (c1min <= ray_tmax);

			// Neither child was intersected => pop stack.

			if (!traverseChild0 && !traverseChild1)   
			{
				nodeAddr = *(int*)stackPtr; // fetch next node by popping the stack 
				stackPtr -= 4; // popping decrements stackPtr by 4 bytes (because stackPtr is a pointer to char)   
			}

			// Otherwise, one or both children intersected => fetch child pointers.

			else  
			{
				// set nodeAddr equal to intersected childnode index (or first childnode when both children are intersected)
				nodeAddr = (traverseChild0) ? cnodes.x : cnodes.y; 

				// Both children were intersected => push the farther one on the stack.

				if (traverseChild0 && traverseChild1) // store closest child in nodeAddr, swap if necessary
				{   
					if (c1min < c0min)  
						swap2(nodeAddr, cnodes.y);  
					stackPtr += 4;  // pushing increments stack by 4 bytes (stackPtr is a pointer to char)
					*(int*)stackPtr = cnodes.y; // push furthest node on the stack
				}
			}

			// First leaf => postpone and continue traversal.
			// leafnodes have a negative index to distinguish them from inner nodes
			// if nodeAddr less than 0 -> nodeAddr is a leaf
			if (nodeAddr < 0 && leafAddr >= 0)  
			{
				//searchingLeaf = false; // required for warp efficiency
				leafAddr = nodeAddr;  
				nodeAddr = *(int*)stackPtr;  // pops next node from stack
				stackPtr -= 4;  // decrements stackptr by 4 bytes (because stackPtr is a pointer to char)
			}

			// All SIMD lanes have found a leaf => process them.

			// to increase efficiency, check if all the threads in a warp have found a leaf before proceeding to the
			// ray/triangle intersection routine
			// this bit of code requires PTX (CUDA assembly) code to work properly

			// if (!__any(searchingLeaf)) -> "__any" keyword: if none of the threads is searching a leaf, in other words
			// if all threads in the warp found a leafnode, then break from while loop and go to triangle intersection

			//if(!__any(leafAddr >= 0))
			//    break;

			// if (!__any(searchingLeaf))
			//	break;    /// break from while loop and go to code below, processing leaf nodes

			// NOTE: inline PTX implementation of "if(!__any(leafAddr >= 0)) break;".
			// tried everything with CUDA 4.2 but always got several redundant instructions.

			unsigned int mask; // replaces searchingLeaf

			asm("{\n"
				"   .reg .pred p;               \n"
				"setp.ge.s32        p, %1, 0;   \n"
				"vote.ballot.b32    %0,p;       \n"
				"}"
				: "=r"(mask)
				: "r"(leafAddr));

			if (!mask)
				break;	
		} 

		
		///////////////////////////////////////////
		/// TRIANGLE INTERSECTION
		//////////////////////////////////////

		// Process postponed leaf nodes.

		while (leafAddr < 0)  /// if leafAddr is negative, it points to an actual leafnode (when positive or 0 it's an innernode)
		{
			// Intersect the ray against each triangle using Sven Woop's algorithm.
			// Woop ray triangle intersection: Woop triangles are unit triangles. Each ray
			// must be transformed to "unit triangle space", before testing for intersection

			for (int triAddr = ~leafAddr;; triAddr += 3)  // triAddr is index in triWoop array (and bitwise complement of leafAddr)
			{ // no defined upper limit for loop, continues until leaf terminator code 0x80000000 is encountered

				// Read first 16 bytes of the triangle.
				// fetch first precomputed triangle edge
				float4 v00 = tex1Dfetch(triWoopTexture, triAddr);
				
				// End marker 0x80000000 (negative zero) => all triangles in leaf processed --> terminate
				if (__float_as_int(v00.x) == 0x80000000) 
					 break;

				// Compute and check intersection t-value (hit distance along ray).
				float Oz = v00.w - origx*v00.x - origy*v00.y - origz*v00.z;   // Origin z
				float invDz = 1.0f / (dirx*v00.x + diry*v00.y + dirz*v00.z);  // inverse Direction z
				float t = Oz * invDz;   
				
				if (t > tmin && t < hitT)
				{
					// Compute and check barycentric u.

					// fetch second precomputed triangle edge
					float4 v11 = tex1Dfetch(triWoopTexture, triAddr + 1);
					float Ox = v11.w + origx*v11.x + origy*v11.y + origz*v11.z;  // Origin.x
					float Dx = dirx * v11.x + diry * v11.y + dirz * v11.z;  // Direction.x
					float u = Ox + t * Dx; /// parametric equation of a ray (intersection point)

					if (u >= 0.0f && u <= 1.0f)
					{
						// Compute and check barycentric v.

						// fetch third precomputed triangle edge
						float4 v22 = tex1Dfetch(triWoopTexture, triAddr + 2);
						float Oy = v22.w + origx*v22.x + origy*v22.y + origz*v22.z;
						float Dy = dirx*v22.x + diry*v22.y + dirz*v22.z;
						float v = Oy + t*Dy;

						if (v >= 0.0f && u + v <= 1.0f)
						{
							// We've got a hit!
							// Record intersection.

							hitT = t;
							hitIndex = triAddr; // store triangle index for shading

							// Closest intersection not required => terminate.
							if (anyHit)  // only true for shadow rays
							{
								nodeAddr = EntrypointSentinel;
								break;
							}

							// compute normal vector by taking the cross product of two edge vectors
							// because of Woop transformation, only one set of vectors works
							
							//trinormal = cross(Vec3f(v22.x, v22.y, v22.z), Vec3f(v11.x, v11.y, v11.z));  // works
							trinormal = cross(Vec3f(v11.x, v11.y, v11.z), Vec3f(v22.x, v22.y, v22.z));
						}
					}
				}
			} // end triangle intersection

			// Another leaf was postponed => process it as well.

			leafAddr = nodeAddr;
			if (nodeAddr < 0)    // nodeAddr is an actual leaf when < 0
			{
				nodeAddr = *(int*)stackPtr;  // pop stack
				stackPtr -= 4;               // decrement with 4 bytes to get the next int (stackPtr is char*)
			}
		} // end leaf/triangle intersection loop
	} // end traversal loop (AABB and triangle intersection)

	// Remap intersected triangle index, and store the result.

	if (hitIndex != -1){
		// hitIndex = tex1Dfetch(triIndicesTexture, hitIndex);
		// remapping tri indices delayed until this point for performance reasons
		// (slow texture memory lookup in de triIndicesTexture) because multiple triangles per node can potentially be hit
	}

	hitTriIdx = hitIndex;
	hitdistance = hitT;
}

// renderKernel:
// - ray scene traversal
// - surface/media interaction
// - return color of a pixel
__device__ Vec3f renderKernel(
	hiprandState* randstate, 
	Vec3f& rayorig, 
	Vec3f& raydir, 
	const Camera* cudaRendercam) 
{
	Vec3f mask = Vec3f(1.0f, 1.0f, 1.0f); // colour mask
	Vec3f accucolor = Vec3f(0.0f, 0.0f, 0.0f); // accumulated colour
	
	int airMedium = MEDIUM_NO;
	int medium = airMedium;
	int objMedium;

	for (int bounces = 0; 
		#if USE_RUSSIAN == true
		hiprand_uniform(randstate) < RUSSIAN_P && bounces < LIGHT_BOUNCE;
		#else
		bounces < LIGHT_BOUNCE; 	
		#endif
		bounces++){

		int hitSphereIdx = -1;
		int hitTriAddr = -1;
		int geomtype = -1;

		float hitSphereDist = 1e20;
		float hitDistance = 1e20;
		float sceneT = 1e20;

		Vec3f objcol = Vec3f(0, 0, 0);
		Vec3f emit = Vec3f(0, 0, 0);

		Vec3f hitpoint; // intersection point
		Vec3f n; // normal
		Vec3f nl; // oriented normal
		Vec3f nextdir; // ray direction of next path segment
		Vec3f trinormal = Vec3f(0, 0, 0);
		Vec2f hitUv = Vec2f(0, 0);

		Refl_t refltype;

		float etaT = 1.43f;

		// ------------------------ scene interaction ----------------------------

		// triangles
		intersectBVHandTriangles(
			make_float4(rayorig.x, rayorig.y, rayorig.z, RAY_MIN), 
			make_float4(raydir.x, raydir.y, raydir.z, RAY_MAX),
			hitTriAddr, 
			hitDistance, 
			trinormal,
			false);
		
		if (hitDistance < sceneT && hitDistance > RAY_MIN) { // triangle hit
			sceneT = hitDistance;
			geomtype = GEO_TRIANGLE;
		}

		// ground
		// GroundPlane ground {-0.78f};
		// if ((hitSphereDist = ground.intersect(Ray(rayorig, raydir)))
		//   && hitSphereDist < sceneT 
		//   && hitSphereDist > RAY_MIN) { 
		// 	sceneT = hitSphereDist;
		// 	geomtype = GEO_GROUND;
		// }

		// spheres
		//Vec3f lightBallPos = Vec3f(8.0f * cosf(cudaRendercam->envMapRotation), 0.0f, 8.0f * sinf(cudaRendercam->envMapRotation));
		Sphere spheres[] = {
			//{ 0.78f, { 0.0f, 0.0f, -3.0f }, { 0.0, 0.0, 0.0 }, { 1.0f, 1.0f, 1.0f }, MAT_GLASS, 1},
			{ 0.0f, { 0.0f, 0.0f, 0.0f }, { 0.0, 0.0, 0.0 }, { 0.0f, 0.0f, 0.0f }, MAT_DIFF, MEDIUM_NO}, // null
			//{ 1.0f, lightBallPos, { 2.0, 2.0, 2.0 }, { 0.0f, 0.0f, 0.0f }, MAT_EMIT, MEDIUM_NO} // light
		};
		float numspheres = sizeof(spheres) / sizeof(Sphere);
		for (int i = int(numspheres); i--;){  // for all spheres in scene
			if ((hitSphereDist = spheres[i].intersect(Ray(rayorig, raydir)))  // keep track of distance from origin to closest intersection point
			&& hitSphereDist < sceneT && hitSphereDist > RAY_MIN) { 
				sceneT = hitSphereDist; 
				hitSphereIdx = i; 
				geomtype = GEO_SPHERE; 
			}
		}

		// participating media
		if (medium != MEDIUM_NO) {
			MediumSS med {{0.74 * 30, 0.88 * 30, 1.01 * 30}, {0.032, 0.17, 0.48}, 0.5f};
			bool sampledMedium;
			HomogeneousMedium(
				hiprand_uniform(randstate), hiprand_uniform(randstate), hiprand_uniform(randstate), hiprand_uniform(randstate),
				mask,
				med.getSigmaT(), med.sigmaS, med.g,
				sceneT,
				rayorig, raydir,
				hitpoint, nextdir,
				sampledMedium
			);
			if (sampledMedium) {
				rayorig = hitpoint;
				raydir = nextdir;
				continue;
			}
		}

		// environmental sphere
		if (sceneT > 1e10f) {
			// Convert (normalized) dir to spherical coordinates.
			float longlatX = atan2f(raydir.x, raydir.z); // Y is up, swap x for y and z for x
			longlatX = longlatX < 0.f ? longlatX + TWO_PI : longlatX;  // wrap around full circle if negative
			float longlatY = acosf(raydir.y); // add RotateMap at some point, see Fragmentarium
			
			float u = fmod(longlatX / (float)TWO_PI + cudaRendercam->envMapRotation, 1.0f); // +offsetY;
			float v = longlatY / M_PI;

			float4 HDRcol = tex2D(HDRtexture, u, v);
			if (cudaRendercam->testLighting) {
				emit = Vec3f(HDRcol.x, HDRcol.y, HDRcol.z) * 2.0f;
			} else {
				emit = Vec3f(0.1f, 0.1f, 0.1f);
			}
			

			accucolor += (mask * emit); 
			return accucolor; 
		}

		// ---------------------- interaction ----------------------
		hitpoint = rayorig + raydir * sceneT;

		// GROUND:
		if (geomtype == GEO_GROUND) {
			n = Vec3f(0,1,0);	// normal
			objcol = Vec3f(0.3f, 0.3f, 0.3f);   // object colour
			emit = Vec3f(0,0,0);  // object emission
			refltype = MAT_DIFF;
			objMedium = MEDIUM_NO;
		}
		// SPHERES:
		else if (geomtype == GEO_SPHERE) {
			Sphere &hitsphere = spheres[hitSphereIdx]; // hit object with closest intersection
			n = hitpoint - hitsphere.pos;	// normal
			objcol = hitsphere.col;   // object colour
			emit = hitsphere.emi;  // object emission
			refltype = hitsphere.refl;
			objMedium = hitsphere.medium;
		}
		// TRIANGLES:
		else if (geomtype == GEO_TRIANGLE) {
			float4 p0 = tex1Dfetch(triDebugTexture, hitTriAddr);
			float4 p1 = tex1Dfetch(triDebugTexture, hitTriAddr + 1);
			float4 p2 = tex1Dfetch(triDebugTexture, hitTriAddr + 2);

			float2 uv0 = tex1Dfetch(triUvTexture, hitTriAddr);
			float2 uv1 = tex1Dfetch(triUvTexture, hitTriAddr + 1);
			float2 uv2 = tex1Dfetch(triUvTexture, hitTriAddr + 2);

			float4 normal0 = tex1Dfetch(triNormalTexture, hitTriAddr);
			float4 normal1 = tex1Dfetch(triNormalTexture, hitTriAddr + 1);
			float4 normal2 = tex1Dfetch(triNormalTexture, hitTriAddr + 2);

			float u, v, w;
			Barycentric(hitpoint, Vec3f(p0.x, p0.y, p0.z), Vec3f(p1.x, p1.y, p1.z), Vec3f(p2.x, p2.y, p2.z), u, v, w);

			hitUv = Vec2f(uv0.x, uv0.y) * u + Vec2f(uv1.x, uv1.y) * v + Vec2f(uv2.x, uv2.y) * w;

			Vec3f smoothNormal = Vec3f(normal0.x, normal0.y, normal0.z) * u
				               + Vec3f(normal1.x, normal1.y, normal1.z) * v
				               + Vec3f(normal2.x, normal2.y, normal2.z) * w;

			if (cudaRendercam->testNormal) {
				n = smoothNormal;
			} else {
				n = trinormal;
			}

			float4 colorTex = tex2D(colorTexture, hitUv.x, hitUv.y); 

			if (cudaRendercam->testTexture) {
				objcol = Vec3f(colorTex.x, colorTex.y, colorTex.z);
			} else {
				objcol = Vec3f(1.0f, 1.0f, 1.0f);
			}
			
			emit = Vec3f(0.0, 0.0, 0.0);

			if (cudaRendercam->testMaterialIdx == 0) { // diff
				refltype = MAT_DIFF;
				objMedium = MEDIUM_NO;
			} else if (cudaRendercam->testMaterialIdx == 1) { // mirror
				refltype = MAT_MIRROR;
				objMedium = MEDIUM_NO;
			} else if (cudaRendercam->testMaterialIdx == 3) { // glass
				refltype = MAT_GLASS;
				objMedium = MEDIUM_NO;
			} else if (cudaRendercam->testMaterialIdx == 4) { // no surface + medium
				refltype = MAT_NO;
				objMedium = MEDIUM_TEST;
			} else if (cudaRendercam->testMaterialIdx == 5) { // glass + medium
				refltype = MAT_GLASS;
				objMedium = MEDIUM_TEST;
			} else {
				refltype = MAT_DIFF;
				objMedium = MEDIUM_NO;
			}
		}

		n.normalize();
		bool into = dot(n, raydir) < 0;
		nl = into ? n : n * -1;

		accucolor += (mask * emit);

		// ------------------------ material ----------------------------
		if (refltype == MAT_EMIT) {
			return accucolor; 
		} else if (refltype == MAT_DIFF) {
			lambertianReflection(hiprand_uniform(randstate), hiprand_uniform(randstate), nextdir, nl);
			hitpoint += nl * RAY_MIN; 
			mask *= objcol;
		} 
		else if (refltype == MAT_MIRROR) {
			nextdir = raydir - n * dot(n, raydir) * 2.0f;
			nextdir.normalize();
			hitpoint += nl * RAY_MIN;
			mask *= objcol;
		}
		else if (refltype == MAT_GLASS) {
			bool refl;
			specularGlass(hiprand_uniform(randstate), into, raydir, nextdir, nl, refl, etaT);
			hitpoint += nl * RAY_MIN * (refl ? 1 : -1);
			if (airMedium != objMedium) medium = (medium == airMedium) ? (refl ? airMedium : objMedium) : (refl ? objMedium : airMedium);
			if (!refl) mask *= objcol;
		} else if (refltype == MAT_NO) {
			bool refl = false;
			nextdir = raydir;
			hitpoint += nl * RAY_MIN * (refl ? 1 : -1);
			if (airMedium != objMedium) medium = (medium == airMedium) ? (refl ? airMedium : objMedium) : (refl ? objMedium : airMedium);
			if (!refl) mask *= objcol;
		}
		// bssrdf

		rayorig = hitpoint; 
		raydir = nextdir; 
	}

	return accucolor;
}

// pathTracingKernel:
// - originate ray of a pixel
// - anti-aliasing
// - depth of field
// - return averaged color of the pixel
__global__ void pathTracingKernel(
	Vec3f* output, 
	Vec3f* accumbuffer, 
	unsigned int framenumber, 
	unsigned int hashedframenumber, 
	const Camera* cudaRendercam)
{
  // assign a CUDA thread to every pixel by using the threadIndex
  unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

  // global threadId, see richiesams blogspot
  int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
  //int pixelx = threadId % scrwidth; // pixel x-coordinate on screen
  //int pixely = threadId / scrwidth; // pixel y-coordintate on screen

  // create random number generator and initialise with hashed frame number, see RichieSams blogspot
  hiprandState randState; // state of the random number generator, to prevent repetition
  hiprand_init(hashedframenumber + threadId, 0, 0, &randState);

  Vec3f finalcol; // final pixel colour 
  finalcol = Vec3f(0.0f, 0.0f, 0.0f); // reset colour to zero for every pixel	
  //Vec3f rendercampos = Vec3f(0, 0.2, 4.6f); 
  Vec3f rendercampos = Vec3f(cudaRendercam->position.x, cudaRendercam->position.y, cudaRendercam->position.z);

  int i = (scrheight - y - 1) * scrwidth + x; // pixel index in buffer	
  int pixelx = x; // pixel x-coordinate on screen
  int pixely = scrheight - y - 1; // pixel y-coordintate on screen

  Vec3f camdir = Vec3f(0, -0.042612, -1); camdir.normalize();
  Vec3f cx = Vec3f(scrwidth * .5135f / scrheight, 0.0f, 0.0f);  // ray direction offset along X-axis 
  Vec3f cy = (cross(cx, camdir)).normalize() * .5135f; // ray dir offset along Y-axis, .5135 is FOV angle

  for (int s = 0; s < NUM_SAMPLE; s++) {

    // compute primary ray direction
    // use camera view of current frame (transformed on CPU side) to create local orthonormal basis
    Vec3f rendercamview = Vec3f(cudaRendercam->view.x, cudaRendercam->view.y, cudaRendercam->view.z); rendercamview.normalize(); // view is already supposed to be normalized, but normalize it explicitly just in case.
    Vec3f rendercamup = Vec3f(cudaRendercam->up.x, cudaRendercam->up.y, cudaRendercam->up.z); rendercamup.normalize();
    Vec3f horizontalAxis = cross(rendercamview, rendercamup); horizontalAxis.normalize(); // Important to normalize!
    Vec3f verticalAxis = cross(horizontalAxis, rendercamview); verticalAxis.normalize(); // verticalAxis is normalized by default, but normalize it explicitly just for good measure.

    Vec3f middle = rendercampos + rendercamview;
    Vec3f horizontal = horizontalAxis * tanf(cudaRendercam->fov.x * 0.5 * (M_PI / 180)); // Treating FOV as the full FOV, not half, so multiplied by 0.5
    Vec3f vertical = verticalAxis * tanf(-cudaRendercam->fov.y * 0.5 * (M_PI / 180)); // Treating FOV as the full FOV, not half, so multiplied by 0.5

    // anti-aliasing
    // calculate center of current pixel and add random number in X and Y dimension
    // based on https://github.com/peterkutz/GPUPathTracer 

    float jitterValueX = hiprand_uniform(&randState) - 0.5;
    float jitterValueY = hiprand_uniform(&randState) - 0.5;
    float sx = (jitterValueX + pixelx) / (cudaRendercam->resolution.x - 1);
    float sy = (jitterValueY + pixely) / (cudaRendercam->resolution.y - 1);

    // compute pixel on screen
    Vec3f pointOnPlaneOneUnitAwayFromEye = middle + (horizontal * ((2 * sx) - 1)) + (vertical * ((2 * sy) - 1));
    Vec3f pointOnImagePlane = rendercampos + ((pointOnPlaneOneUnitAwayFromEye - rendercampos) * cudaRendercam->focalDistance); // Important for depth of field!		

    // calculation of depth of field / camera aperture 
    // based on https://github.com/peterkutz/GPUPathTracer 

    Vec3f aperturePoint = Vec3f(0, 0, 0);

    if (cudaRendercam->apertureRadius > 0.00001) { // the small number is an epsilon value.

      // generate random numbers for sampling a point on the aperture
      float random1 = hiprand_uniform(&randState);
      float random2 = hiprand_uniform(&randState);

      // randomly pick a point on the circular aperture
      float angle = TWO_PI * random1;
      float distance = cudaRendercam->apertureRadius * sqrtf(random2);
      float apertureX = cos(angle) * distance;
      float apertureY = sin(angle) * distance;

      aperturePoint = rendercampos + (horizontalAxis * apertureX) + (verticalAxis * apertureY);
		}
		else { // zero aperture
      aperturePoint = rendercampos;
    }

    // calculate ray direction of next ray in path
    Vec3f apertureToImagePlane = pointOnImagePlane - aperturePoint;
    apertureToImagePlane.normalize(); // ray direction needs to be normalised

    // ray direction
    Vec3f rayInWorldSpace = apertureToImagePlane;
    rayInWorldSpace.normalize();

    // ray origin
    Vec3f originInWorldSpace = aperturePoint;

    finalcol += renderKernel(
		&randState, 
        originInWorldSpace, 
		rayInWorldSpace,
		cudaRendercam) * (1.0f / NUM_SAMPLE);
	}

	// add pixel colour to accumulation buffer (accumulates all samples) 
	accumbuffer[i] += finalcol;

	// averaged colour: divide colour by the number of calculated frames so far
	Vec3f tempcol = accumbuffer[i] / framenumber;

	// union struct required for mapping pixel colours to OpenGL buffer
	union Colour  // 4 bytes = 4 chars = 1 float
	{
		float c;
		uchar4 components;
	};

	Colour fcolour;
	Vec3f colour = Vec3f(clamp(tempcol.x, 0.0f, 1.0f), clamp(tempcol.y, 0.0f, 1.0f), clamp(tempcol.z, 0.0f, 1.0f));

	// convert from 96-bit to 24-bit colour + perform gamma correction
	fcolour.components = make_uchar4((unsigned char)(powf(colour.x, 1 / 2.2f) * 255),
		(unsigned char)(powf(colour.y, 1 / 2.2f) * 255),
		(unsigned char)(powf(colour.z, 1 / 2.2f) * 255), 1);

	// store pixel coordinates and pixelcolour in OpenGL readable outputbuffer
	output[i] = Vec3f(x, y, fcolour.c);
}

// cudaRender
// - bind buffers to textures
// - kernal dimension setting
// - launch kernal
void cudaRender(const float4* nodes, const float4* triWoops, const float4* debugTris, const int* triInds, 
	Vec3f* outputbuf, Vec3f* accumbuf, const hipArray* HDRmap, const hipArray* colorArray, const unsigned int framenumber, const unsigned int hashedframenumber, 
	const unsigned int nodeSize, const unsigned int leafnodecnt, const unsigned int tricnt, const Camera* cudaRenderCam, const float2 *cudaUvPtr,
	const float4 *cudaNormalPtr)
{
	static bool firstTime = true;

	// texture binding
	if (firstTime) {
		firstTime = false;
		
		// bvh textures
		hipChannelFormatDesc channel0desc = hipCreateChannelDesc<int>();
		hipBindTexture(NULL, &triIndicesTexture, triInds, &channel0desc, (tricnt * 3 + leafnodecnt) * sizeof(int));

		hipChannelFormatDesc channel1desc = hipCreateChannelDesc<float4>();
		hipBindTexture(NULL, &triWoopTexture, triWoops, &channel1desc, (tricnt * 3 + leafnodecnt) * sizeof(float4));

		hipChannelFormatDesc channel2desc = hipCreateChannelDesc<float2>();
		hipBindTexture(NULL, &triUvTexture, cudaUvPtr, &channel2desc, (tricnt * 3 + leafnodecnt) * sizeof(float2));

		hipChannelFormatDesc channel3desc = hipCreateChannelDesc<float4>();
		hipBindTexture(NULL, &triDebugTexture, debugTris, &channel3desc, (tricnt * 3 + leafnodecnt) * sizeof(float4));

		hipChannelFormatDesc channel4desc = hipCreateChannelDesc<float4>();
		hipBindTexture(NULL, &bvhNodesTexture, nodes, &channel4desc, nodeSize * sizeof(float4)); 

		hipChannelFormatDesc channel5desc = hipCreateChannelDesc<float4>();
		hipBindTexture(NULL, &triNormalTexture, cudaNormalPtr, &channel5desc, (tricnt * 3 + leafnodecnt) * sizeof(float4)); 

		// hdr texture
		HDRtexture.addressMode[0] = hipAddressModeClamp;
		HDRtexture.addressMode[1] = hipAddressModeClamp;
		HDRtexture.filterMode = hipFilterModeLinear;
		HDRtexture.normalized = true;

		hipChannelFormatDesc channel6desc = hipCreateChannelDesc<float4>(); 
		hipBindTextureToArray(HDRtexture, HDRmap, channel6desc);

		// color texture
		colorTexture.normalized = true;
		colorTexture.filterMode = hipFilterModeLinear;
		colorTexture.addressMode[0] = hipAddressModeWrap;
		colorTexture.addressMode[1] = hipAddressModeWrap;
		colorTexture.maxAnisotropy = 8;
		colorTexture.sRGB = true;

		hipChannelFormatDesc channel7desc = hipCreateChannelDesc<float4>(); 
		hipBindTextureToArray(colorTexture, colorArray, channel7desc);

		printf("CudaWoopTriangles texture initialised, tri count: %d\n", tricnt);
	}

	dim3 threadsPerBlock (16, 16, 1);   // dim3 CUDA specific syntax, block and grid are required to schedule CUDA threads over streaming multiprocessors
	dim3 fullBlocksPerGrid (scrwidth / threadsPerBlock.x, scrheight / threadsPerBlock.y, 1);

	// Configure grid and block sizes:
	// int threadsPerBlock = 256;
	// Compute the number of blocks required, performing a ceiling operation to make sure there are enough:
	// int fullBlocksPerGrid = ((scrwidth * scrheight) + threadsPerBlock - 1) / threadsPerBlock;
	// <<<fullBlocksPerGrid, threadsPerBlock>>>
	pathTracingKernel <<< fullBlocksPerGrid, threadsPerBlock >>> (
		outputbuf, 
		accumbuf, 
		framenumber, 
		hashedframenumber, 
		cudaRenderCam);  // texdata, texoffsets

}
